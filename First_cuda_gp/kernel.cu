#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <fstream>
#include "book.h"
#include "gridcheck.h"
using namespace std;
# define Section 12  // number of cooling sections
# define CoolSection 8
# define MoldSection 4

float ccml[Section + 1] = { 0.0,0.2,0.4,0.6,0.8,1.0925,2.27,4.29,5.831,9.6065,13.6090,19.87014,28.599 }; // The cooling sections
float H_Init[Section] = { 1380,1170,980,800,1223.16,735.05,424.32,392.83,328.94,281.64,246.16,160.96 };  // The heat transfer coefficients in the cooling sections
float H_Init_Temp[Section] = { 1380,1170,980,800,1223.16,735.05,424.32,392.83,328.94,281.64,246.16,160.96 };  // The heat transfer coefficients in the cooling sections
float Taim[CoolSection] = { 966.149841, 925.864746, 952.322083, 932.175537, 914.607117, 890.494263, 870.804443, 890.595825 };
float *Calculation_MeanTemperature(int nx, int ny, int nz, float dy, float *ccml, float *T);
hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, int num_blocks, int num_threadsx, int num_threadsy);
__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd);
__device__ float Boundary_Condition(int j, float dx, float *ccml_zone, float *H_Init);

__global__ void addKernel(float *T_New, float *T_Last, float *ccml, float *H_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, bool disout)
{
	int i = threadIdx.x;
	int m = threadIdx.y;
	int j = blockIdx.x;
	int idx = j * nx * nz + m * nx + i;
	int ND = nx * nz;
	int D = nx;

	float pho, Ce, lamd; // physical parameters pho represents desity, Ce is specific heat and lamd is thermal conductivity
	float a, T_Up, T_Down, T_Right, T_Left, T_Forw, T_Back, h = 100.0, Tw = 30.0, Vcast = -0.02, T_Cast = 1558.0;

	if (disout) {
		Physicial_Parameters(T_Last[idx], &pho, &Ce, &lamd);
		a = (lamd) / (pho*Ce);
		h = Boundary_Condition(j, dy, ccml, H_Init);
		if (j == 0) //1
		{
			T_New[idx] = T_Cast;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = (a*tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ (a*tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == 0)  //15
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else  //27
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}
	}

	else
	{
		Physicial_Parameters(T_New[idx], &pho, &Ce, &lamd);
		a = (lamd) / (pho*Ce);
		h = Boundary_Condition(j, dy, ccml, H_Init);
		if (j == 0) //1
		{
			T_Last[idx] = T_Cast;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == 0)  //15
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else  //27
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}
	}
}
int main()
{
	const int nx = 21, ny = 3000, nz = 21;   // nx is the number of grid in x direction, ny is the number of grid in y direction.
	int num_blocks = 1, num_threadsx = 1, num_threadsy = 1;// num_threadsz = 1; // block number(1D)  thread number in x and y dimension(2D)
	int tnpts = 10001;  // time step
	float T_Cast = 1558.0, Lx = 0.25, Ly = 28.599, Lz = 0.25, t_final = 2000.0, dx, dy, dz, tao;  // T_Cast is the casting temperature Lx and Ly is the thick and length of steel billets
	float *T_Init;

	T_Init = (float *)calloc(nx * ny * nz, sizeof(float));  // Initial condition
	num_threadsx = nx;
	num_threadsy = nz;
	num_blocks = ny;

	for (int m = 0; m < nz; m++)
		for (int j = 0; j < ny; j++)
	       for (int i = 0; i < nx; i++)
			   T_Init[nx * ny * m + j * nx + i] = T_Cast;  // give the initial condition

	dx = Lx / (nx - 1);            // the grid size x
	dy = Ly / (ny - 1);            // the grid size y
	dz = Lz / (nz - 1);            // the grid size y
	tao = t_final / (tnpts - 1);   // the time step size
	//gridcheck(dx, dy, tao);

	cout << "Casting Temperature " << T_Cast << endl;
	cout << "The length of steel billets(m) " << Ly << endl;
	cout << "The width of steel billets(m) " << Lz << endl;
	cout << "The thick of steel billets(m) " << Lx << endl;
	cout << "dx(m) " << dx << ", ";
	cout << "dy(m) " << dy << ", ";
	cout << "dz(m) " << dz << ", ";
	cout << "tao(s) " << tao << ", ";
	cout << "simulation time(s) " << t_final << endl;

	clock_t timestart = clock();
	hipError_t cudaStatus = addWithCuda(T_Init, dx, dy, dz, tao, nx, ny, nz, tnpts, num_blocks, num_threadsx, num_threadsy);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	clock_t timeend = clock();

	cout << "running time = " << (timeend - timestart);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, int num_blocks, int num_threadsx, int num_threadsy)
{
	float *dev_T_New, *dev_T_Last, *dev_ccml, *dev_H_Init; // the point on GPU
	float *T_Result, *Delta_H_Init, *T_HoldLast, **Mean_TSurfaceElement, **Mean_TSurfaceElementOne, **JacobianMatrix;
	float dh = 10.0, arf1, arf2, step = -0.0001;
	const int Num_Iter = 10, PrintLabel = 0;                         // The result can be obtained by every Num_Iter time step
	volatile bool dstOut = true;

	T_Result = (float *)calloc(nx * ny * nz, sizeof(float)); // The temperature of steel billets
	Delta_H_Init = (float*)calloc(CoolSection, sizeof(float));

	T_HoldLast = (float*)calloc(nz * ny * nx, sizeof(float));

	JacobianMatrix = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		JacobianMatrix[i] = (float*)calloc(CoolSection, sizeof(float));

	Mean_TSurfaceElement = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		Mean_TSurfaceElement[i] = (float*)calloc(CoolSection, sizeof(float));

	Mean_TSurfaceElementOne = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		Mean_TSurfaceElementOne[i] = (float*)calloc(CoolSection, sizeof(float));

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(0));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_ccml, (Section + 1) * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init, Section * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_T_Last, T_Init, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_ccml, ccml, (Section + 1) * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init, Section * sizeof(float), hipMemcpyHostToDevice));

	dim3 threadsPerBlock(num_threadsx, num_threadsy);

	for (int i = 0; i < tnpts; i++)
	{
		if (i % Num_Iter == 0)
		{
			HANDLE_ERROR(hipMemcpy(T_HoldLast, dev_T_Last, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
			for (int m = 0; m < CoolSection + 1; m++)
			{
				if (m == CoolSection)
				{
					for (int temp = 0; temp < Section; temp++)
						H_Init_Temp[temp] = H_Init[temp];
					HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init_Temp, Section * sizeof(float), hipMemcpyHostToDevice));
					for (int PNum = 0; PNum < Num_Iter; PNum++)
					{
						addKernel << <num_blocks, threadsPerBlock >> >(dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut);
						dstOut = !dstOut;
					}

					HANDLE_ERROR(hipMemcpy(T_Result, dev_T_New, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
					float* Mean_TSurface = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result);  // calculation the mean surface temperature of steel billets in every cooling sections
					for (int temp = 0; temp < CoolSection; temp++)
						for (int column = 0; column < CoolSection; column++)
							Mean_TSurfaceElementOne[temp][column] = Mean_TSurface[column + MoldSection];
				}

				else
				{
					for (int temp = 0; temp < Section; temp++)
						H_Init_Temp[temp] = H_Init[temp];
					H_Init_Temp[m + MoldSection] = H_Init[m + MoldSection] + dh;
					HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init_Temp, Section * sizeof(float), hipMemcpyHostToDevice));

					for (int PNum = 0; PNum < Num_Iter; PNum++)
					{
						addKernel << <num_blocks, threadsPerBlock >> >(dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut);
						dstOut = !dstOut;
					}

					HANDLE_ERROR(hipMemcpy(T_Result, dev_T_New, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
					float* Mean_TSurface = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result); // calculation the mean surface temperature of steel billets in every cooling sections
					for (int column = 0; column < CoolSection; column++)
						Mean_TSurfaceElement[m][column] = Mean_TSurface[column + MoldSection];
				}
				HANDLE_ERROR(hipMemcpy(dev_T_Last, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));
			}

			for (int row = 0; row < CoolSection; row++)
				for (int column = 0; column < CoolSection; column++)
					JacobianMatrix[row][column] = (Mean_TSurfaceElement[row][column] - Mean_TSurfaceElementOne[row][column]) / dh;

			for (int temp = 0; temp < CoolSection; temp++)
				Delta_H_Init[temp] = 0.0;

			for (int temp = 0; temp < CoolSection; temp++)
				for (int column = 0; column < CoolSection; column++)
					Delta_H_Init[temp] += (Mean_TSurfaceElementOne[temp][column] - Taim[column]) * JacobianMatrix[temp][column];
				

			arf1 = 0.0, arf2 = 0.0;
			for (int temp = 0; temp < CoolSection; temp++)
			{
				for (int column = 0; column < CoolSection; column++)
				{
					arf1 += (Mean_TSurfaceElementOne[0][temp] - Taim[temp]) * JacobianMatrix[temp][column] * Delta_H_Init[column];
					arf2 += JacobianMatrix[temp][column] * Delta_H_Init[column] * JacobianMatrix[temp][column] * Delta_H_Init[column];
				}
			}
			step = -arf1 / ((arf2) + 0.001);

			for (int temp = 0; temp < CoolSection; temp++)
				H_Init[temp + MoldSection] += step *(Delta_H_Init[temp]);
		}

		for (int temp = 0; temp < Section; temp++)
			H_Init_Temp[temp] = H_Init[temp];
		HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init_Temp, Section * sizeof(float), hipMemcpyHostToDevice));
		addKernel << <num_blocks, threadsPerBlock >> >(dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut);
		dstOut = !dstOut;

		if (i % (10 * Num_Iter) == 0)
		{
			HANDLE_ERROR(hipMemcpy(T_Result, dev_T_Last, nx * ny * nz* sizeof(float), hipMemcpyDeviceToHost));
			float* Mean_TSurface = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result);  // calculation the mean surface temperature of steel billets in every cooling sections
		
				cout << "time_step = " << i <<",  "<< "simulation time = " << i * tao;
				cout << endl << "TSurface = " << endl;
				for (int temp = 0; temp < CoolSection; temp++)
					cout << Mean_TSurface[temp + MoldSection] << ", ";

				cout << endl << "TSurface - Taim = " << endl;
				for (int temp = 0; temp < CoolSection; temp++)
					cout << (Mean_TSurface[temp + MoldSection] - Taim[temp]) << ", ";
		}
	}

	    
	ofstream fout;
		fout.open("D:\\Temperature3DGPUMPC_Static.txt");
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				for (int m = 0; m < nz; m++)
					fout << T_Result[nx * nz * j + i * nz + m] << ", ";
				fout << endl;
			}
			fout << endl;
		}
		fout.close();

		fout.open("D:\\SurfaceTemperature3DGPUMPC_Static.txt");
		for (int j = 0; j < ny; j++)
		{
			fout << T_Result[nx * nz * j + 0 * nz + int((nx - 1) / 2)] << ", ";
			fout << endl;
		}
		fout.close();
	

	// Check for any errors launching the kernel
	HANDLE_ERROR(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.


Error:
	hipFree(dev_T_New);
	hipFree(dev_T_Last);
	hipFree(dev_ccml);
	hipFree(dev_H_Init);

	return cudaStatus;
}
// Helper function for using CUDA to add vectors in parallel.

__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd)
{
	float Ts = 1462.0, Tl = 1518.0, lamds = 30, lamdl = 50, phos = 7000, phol = 7500, ce = 540.0, L = 265600.0, fs = 0.0;
	if (T<Ts)
	{
		fs = 0;
		*pho = phos;
		*lamd = lamds;
		*Ce = ce;
	}

	if (T >= Ts&&T <= Tl)
	{
		fs = (T - Ts) / (Tl - Ts);
		*pho = fs*phos + (1 - fs)*phol;
		*lamd = fs*lamds + (1 - fs)*lamdl;
		*Ce = ce + L / (Tl - Ts);
	}

	if (T>Tl)
	{
		fs = 1;
		*pho = phol;
		*lamd = lamdl;
		*Ce = ce;
	}

}

__device__ float Boundary_Condition(int j, float dy, float *ccml_zone, float *H_Init)
{
	float YLabel, h = 0.0;
	YLabel = j*dy;

	for (int i = 0; i < Section; i++)
	{
		if (YLabel >= *(ccml_zone + i) && YLabel <= *(ccml_zone + i + 1))
			h = *(H_Init + i);
	}
	return h;
}

float* Calculation_MeanTemperature(int nx, int ny, int nz, float dy, float *ccml, float *T)
{
	float y;
	int count = 0;
	int i = 0;
	
	float* Mean_TSurface;
	Mean_TSurface = new float[Section];
	for (int i = 0; i < Section; i++)
	{
		Mean_TSurface[i] = 0.0;
		for (int j = 0; j < ny; j++)
		{
			y = j * dy;
			if (y > *(ccml + i) && y <= *(ccml + i + 1))
			{
				Mean_TSurface[i] = Mean_TSurface[i] + T[nx * nz * j + 0 * nz + int((nx - 1) / 2)];
				count++;
			}
		}
		Mean_TSurface[i] = Mean_TSurface[i] / float(count);
		count = 0;
	}
	return Mean_TSurface;
}